#include "cuda_util.cuh"

/*
NVIDIA GPU COMPATIBILITY
*/
void getGrid(unsigned long numElements, dim3 &grid, int device){
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
    
  grid = {(unsigned int)prop.maxGridSize[0],(unsigned int)prop.maxGridSize[1],(unsigned int)prop.maxGridSize[2]};
  if(numElements < grid.x){
    grid.x = numElements;
    grid.y = 1;
    grid.z = 1;
  }
  else{
    grid.x = 65536;
    if(numElements < grid.x*grid.y){
      grid.y = numElements/grid.x;
      grid.y++;
      grid.z = 1;
    }
    else if(numElements < grid.x*grid.y*grid.z){
      grid.z = numElements/(grid.x*grid.y);
      grid.z++;
    }
  }
}
void checkDims(dim3 grid, dim3 block, int device){
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  bool goodDims = true;
  if(grid.x > prop.maxGridSize[0]){
    goodDims = false;
  }
  else if(grid.y > prop.maxGridSize[1]){
    goodDims = false;
  }
  else if(grid.z > prop.maxGridSize[2]){
    goodDims = false;
  }
  else if(block.x > prop.maxThreadsDim[0]){
    goodDims = false;
  }
  else if(block.y > prop.maxThreadsDim[1]){
    goodDims = false;
  }
  else if(block.z > prop.maxThreadsDim[2]){
    goodDims = false;
  }
  else if(block.x*block.y*block.z > prop.maxThreadsPerBlock){
    goodDims = false;
  }
  if(!goodDims){
    std::cerr<<"ERROR: grid or block dims are invalid for given device"<<std::endl;
    exit(-1);
    //TODO replace with exception and make more specific
    //maybe make macro like CudaSafeCall()
  }
}
__host__ void cusolverCheckError(hipsolverStatus_t cusolver_status){
  switch (cusolver_status){
      case HIPSOLVER_STATUS_SUCCESS:
          std::cout<<"CUSOLVER_SUCCESS"<<std::endl;
          break;

      case HIPSOLVER_STATUS_NOT_INITIALIZED:
          std::cout<<"HIPSOLVER_STATUS_NOT_INITIALIZED"<<std::endl;
          exit(-1);

      case HIPSOLVER_STATUS_ALLOC_FAILED:
          std::cout<<"HIPSOLVER_STATUS_ALLOC_FAILED"<<std::endl;
          exit(-1);

      case HIPSOLVER_STATUS_INVALID_VALUE:
          std::cout<<"HIPSOLVER_STATUS_INVALID_VALUE"<<std::endl;
          exit(-1);

      case HIPSOLVER_STATUS_ARCH_MISMATCH:
          std::cout<<"HIPSOLVER_STATUS_ARCH_MISMATCH"<<std::endl;
          exit(-1);

      case HIPSOLVER_STATUS_EXECUTION_FAILED:
          std::cout<<"HIPSOLVER_STATUS_EXECUTION_FAILED"<<std::endl;
          exit(-1);

      case HIPSOLVER_STATUS_INTERNAL_ERROR:
          std::cout<<"HIPSOLVER_STATUS_INTERNAL_ERROR"<<std::endl;
          exit(-1);

      case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
          std::cout<<"HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED"<<std::endl;
          exit(-1);
  }
}
//prints tx2 info relevant to cuda devlopment
//citation: this method comes from the nvidia formums, and has been modified slightly
void printDeviceProperties(){
	std::cout<<"\n---------------START OF DEVICE PROPERTIES---------------\n"<<std::endl;

  int nDevices;
  hipGetDeviceCount(&nDevices);      //find num of devices on tx2

  for (int i = 0; i < nDevices; i++)  //print info on each device
	{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);

    printf("Device Number: %d\n", i);
    printf(" -Device name: %s\n\n", prop.name);
    printf(" -Memory\n  -Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  -Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
    printf("  -Peak Memory Bandwidth (GB/s): %f\n",2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    printf("  -Total Global Memory (bytes): %lo\n", prop.totalGlobalMem);
    printf("  -Total Const Memory (bytes): %lo\n", prop.totalConstMem);
    printf("  -Max pitch allowed for memcpy in regions allocated by hipMallocPitch() (bytes): %lo\n\n", prop.memPitch);
    printf("  -Shared Memory per block (bytes): %lo\n", prop.sharedMemPerBlock);
    printf("  -Max number of threads per block: %d\n",prop.maxThreadsPerBlock);
    printf("  -Max number of blocks: %dx%dx%d\n",prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("  -32bit Registers per block: %d\n", prop.regsPerBlock);
    printf("  -Threads per warp: %d\n\n", prop.warpSize);
    printf("  -Max Threads per Multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  -Total number of Multiprocessors: %d\n",prop.multiProcessorCount);
    printf("  -Shared Memory per Multiprocessor (bytes): %lo\n",prop.sharedMemPerMultiprocessor);
    printf("  -32bit Registers per Multiprocessor: %d\n", prop.regsPerMultiprocessor);
    printf("  -Number of asynchronous engines: %d\n", prop.asyncEngineCount);
    printf("  -Texture alignment requirement (bytes): %lo\n  -Texture base addresses that are aligned to "
    "textureAlignment bytes do not need an offset applied to texture fetches.\n\n", prop.textureAlignment);
    printf(" -Device Compute Capability:\n  -Major revision #: %d\n  -Minor revision #: %d\n", prop.major, prop.minor);

		printf(" -Run time limit for kernels that get executed on this device: ");
		if(prop.kernelExecTimeoutEnabled)
		{
      printf("YES\n");
    }
    else
		{
      printf("NO\n");
    }

    printf(" -Device is ");
    if(prop.integrated)
		{
      printf("integrated. (motherboard)\n");
    }
    else
		{
      printf("discrete. (card)\n\n");
    }

    if(prop.isMultiGpuBoard)
		{
      printf(" -Device is on a MultiGPU configurations.\n\n");
    }

    switch(prop.computeMode)
		{
      case(0):
        printf(" -Default compute mode (Multiple threads can use hipSetDevice() with this device)\n");
        break;
      case(1):
        printf(" -Compute-exclusive-thread mode (Only one thread in one processwill be able to use\n hipSetDevice() with this device)\n");
        break;
      case(2):
        printf(" -Compute-prohibited mode (No threads can use hipSetDevice() with this device)\n");
        break;
      case(3):
        printf(" -Compute-exclusive-process mode (Many threads in one process will be able to use\n hipSetDevice() with this device)\n");
        break;
      default:
        printf(" -GPU in unknown compute mode.\n");
        break;
    }

    if(prop.canMapHostMemory)
		{
      printf("\n -The device can map host memory into the CUDA address space for use with\n hipHostAlloc() or hipHostGetDevicePointer().\n\n");
    }
    else
		{
      printf("\n -The device CANNOT map host memory into the CUDA address space.\n\n");
    }

    printf(" -ECC support: ");
    if(prop.ECCEnabled)
		{
      printf(" ON\n");
    }
    else
		{
      printf(" OFF\n");
    }

    printf(" -PCI Bus ID: %d\n", prop.pciBusID);
    printf(" -PCI Domain ID: %d\n", prop.pciDomainID);
    printf(" -PCI Device (slot) ID: %d\n", prop.pciDeviceID);

    printf(" -Using a TCC Driver: ");
    if(prop.tccDriver)
		{
      printf("YES\n");
    }
    else
		{
      printf("NO\n");
    }
  }
  std::cout<<"\n----------------END OF DEVICE PROPERTIES----------------\n"<<std::endl;
}

/*
SIMPLE CUDA METHODS
*/
__device__ __host__ void printBits(size_t const size, void const * const ptr){
  unsigned char *b = (unsigned char*) ptr;
  unsigned char byte;
  int i, j;
  printf("bits - ");
  for (i=size-1;i>=0;i--){
    for (j=7;j>=0;j--){
      byte = (b[i] >> j) & 1;
      printf("%u", byte);
    }
  }
  printf("\n");
}